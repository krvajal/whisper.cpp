#include "ggml.h"
#include "stdio.h"

struct ggml_context {
    size_t mem_size;
    void* mem_buffer;
};

ggml_context * ggml_init(ggml_init_params params){
    
    void* mem_buffer = NULL;
    hipError_t err = hipSuccess;
    err = hipMalloc((void**)&mem_buffer, params.mem_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    }

    ggml_context* ctx = (ggml_context*)malloc(sizeof(struct ggml_context));
    ctx -> mem_size = params.mem_size;
    ctx -> mem_buffer = mem_buffer;

    return ctx;
}

ggml_tensor* ggml_new_tensor_1d(
        struct ggml_context * ctx,
        enum   ggml_type type,
        int    ne0)  {
             return ggml_new_tensor(ctx, type, 1, &ne0);
}

ggml_tensor* ggml_new_tensor(
        struct ggml_context * ctx,
        enum   ggml_type type,
        int    n_dims,
        const int *ne) {
    return NULL;
}



float ggml_get_f32_1d(const struct ggml_tensor * tensor, int i){
    return 1.0f;
}

void  ggml_set_f32_1d(const struct ggml_tensor * tensor, int i, float value){

}